#include "hip/hip_runtime.h"
#include "SigmoidLayer.cuh"
#include "Matrix.cuh"
#include <cmath>

//Kernel and device functions to be run on GPU
//sigmoid
__device__ float sigmoid(float z) {
    return (1.0 / (1.0 + std::exp(-1.0 * z)));
}

//sigmoid prime
__device__ float sigmoidPrime(float z) {
    float a = (1.0 + std::exp(-1.0 * z));
    return (std::exp(-1.0 * z)) / (pow(a, 2));
}

__global__ void getActivation(float* w, float* x, float* a, float* b, float* z, int xDim, int yDim) {
    int rowIndexW = (threadIdx.x + blockDim.x * blockIdx.x); //only doing 1D thread blocks because of matrix multiplication
    int stride = gridDim.x * blockDim.x;

    //W*x
    for (int i = rowIndexW; i < yDim; i = i + stride) {
        a[i] = 0; //reinitialize to 0
        for (int j = 0; j < xDim; j++) {
            a[i] += w[(i * xDim) + j] * x[j];
        }
        //+b
        a[i] += b[i];
        //Sigmoid(Z)
        z[i] = a[i]; //set weighted input matrix before applying activation function
        a[i] = sigmoid(a[i]);
    }

}

//used for backpropagation
__global__ void backPropError(float* nextError, float* w, float* z, float* error, int xDim, int yDim) {
    int rowIndex = threadIdx.x + (blockIdx.x * blockDim.x);
    int stride = blockDim.x * gridDim.x;

    //i < xDim and j < yDim because multiplying transpose of next layer's weight matrix
    for (int i = rowIndex; i < xDim; i = i + stride) {
        error[i] = 0; //reinitialize to 0
        for (int j = 0; j < yDim; j++) {
            error[i] += w[(j * xDim) + i] * nextError[j];
        }
        //hadamard product with sigmoid prime w.r.t weighted input
        error[i] = error[i] * sigmoidPrime(z[i]);
    }
}

//a = activation pointer, b = activationPrime pointer
__global__ void getFunctionPointers(Layer::act* a, Layer::act* b) {
    *a = sigmoid;
    *b = sigmoidPrime;
}

//constructor
SigmoidLayer::SigmoidLayer(int prevNumNeurons, int numNeurons) : Layer{prevNumNeurons, numNeurons} {
    //called after base class (Layer) constructor called, so device memory already allocated
    getFunctionPointers<<<1,1>>>(this->activation.get(), this->activationPrime.get());
}

void SigmoidLayer::callGetActivation(dim3 blocks, dim3 threads, float* w, float* x, float* a, float* b, float* z, int xDim, int yDim) {
    getActivation<<<blocks, threads>>>(w, x, a, b, z, xDim, yDim);
}

void SigmoidLayer::callBackPropError(dim3 blocks, dim3 threads, float* nextError, float* w, float* z, float* error, int xDim, int yDim) {
    backPropError<<<blocks, threads>>>(nextError, w, z, error, xDim, yDim);
}
