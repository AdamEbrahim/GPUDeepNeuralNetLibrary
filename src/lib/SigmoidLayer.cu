#include "hip/hip_runtime.h"
#include "SigmoidLayer.cuh"
#include "Matrix.cuh"
#include "utils.cuh"
#include <cmath>

//Kernel and device functions to be run on GPU

__global__ void getActivation(float* w, float* x, float* a, float* b, float* z, int xDim, int yDim) {
    int rowIndexW = (threadIdx.x + blockDim.x * blockIdx.x); //only doing 1D thread blocks because of matrix multiplication
    int stride = gridDim.x * blockDim.x;

    //W*x
    for (int i = rowIndexW; i < yDim; i = i + stride) {
        a[i] = 0; //reinitialize to 0
        for (int j = 0; j < xDim; j++) {
            a[i] += w[(i * xDim) + j] * x[j];
        }
        //+b
        a[i] += b[i];
        //Sigmoid(Z)
        z[i] = a[i]; //set weighted input matrix before applying activation function
        a[i] = sigmoid(a[i]);
    }

}

//used for backpropagation
__global__ void backPropError(float* nextError, float* w, float* z, float* error, int xDim, int yDim) {
    int rowIndex = threadIdx.x + (blockIdx.x * blockDim.x);
    int stride = blockDim.x * gridDim.x;

    //i < xDim and j < yDim because multiplying transpose of next layer's weight matrix
    for (int i = rowIndex; i < xDim; i = i + stride) {
        error[i] = 0; //reinitialize to 0
        for (int j = 0; j < yDim; j++) {
            error[i] += w[(j * xDim) + i] * nextError[j];
        }
        //hadamard product with sigmoid prime w.r.t weighted input
        error[i] = error[i] * sigmoidPrime(z[i]);
    }
}

//constructor
SigmoidLayer::SigmoidLayer(int prevNumNeurons, int numNeurons) : Layer{prevNumNeurons, numNeurons} {

}

void SigmoidLayer::callGetActivation(dim3 blocks, dim3 threads, float* w, float* x, float* a, float* b, float* z, int xDim, int yDim) {
    getActivation<<<blocks, threads>>>(w, x, a, b, z, xDim, yDim);
}

void SigmoidLayer::callBackPropError(dim3 blocks, dim3 threads, float* nextError, float* w, float* z, float* error, int xDim, int yDim) {
    backPropError<<<blocks, threads>>>(nextError, w, z, error, xDim, yDim);
}
