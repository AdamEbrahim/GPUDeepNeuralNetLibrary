#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <cmath>

//sigmoid
__device__ float sigmoid(float z) {
    return (1.0 / (1.0 + std::exp(-1.0 * z)));
}

//sigmoid prime
__device__ float sigmoidPrime(float z) {
    float a = (1.0 + std::exp(-1.0 * z))
    return (std::exp(-1.0 * z)) / (pow(a, 2));
}