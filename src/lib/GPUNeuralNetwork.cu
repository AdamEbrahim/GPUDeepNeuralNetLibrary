#include "GPUNeuralNetwork.cuh"
#include "SigmoidLayer.cuh"
#include "Matrix.cuh"
#include <algorithm>
#include <random>
#include <numeric>

GPUNeuralNetwork::GPUNeuralNetwork(std::string costFunc, int inputLayerNeurons, int numOutputClasses, float learningRate) try : costFunction(costFunc), numInputLayerNeurons{inputLayerNeurons}, numOutputClasses{numOutputClasses}, learningRate{learningRate}, inputActivations{1, inputLayerNeurons}, trueOutput{1, numOutputClasses} 
    {
        //allocate host and cuda memory for input layer activations
        inputActivations.allocateHostMemory();
        inputActivations.allocateCUDAMemory();
        //allocate host and cuda memory for true output vector
        trueOutput.allocateHostMemory();
        trueOutput.allocateCUDAMemory();
    }
    catch (std::string type) {
        std::cout << "Invalid cost function: " << type << std::endl;
    }

GPUNeuralNetwork::~GPUNeuralNetwork() { //destructor
    for (int i = 0; i < this->layers.size(); i++) {
        delete this->layers[i];
    }
}

//final layer count will no matter what be same as numOutputClasses, no matter what is given
void GPUNeuralNetwork::initializeLayers(std::vector<std::string> layerTypes, std::vector<int> layerCounts) {
    try {
        if (layerTypes.size() != layerCounts.size()) {
            throw(layerTypes.size());
        }

        int prevLayerNeurons = 0;
        for (int i = 0; i < layerTypes.size(); i++) {
            
            if (i == 0) {
                prevLayerNeurons = this->numInputLayerNeurons;
            } else {
                prevLayerNeurons = layerCounts[i-1];
            }

            Layer* createdLayer;
            //final layer count will no matter what be same as numOutputClasses, no matter what is given
            int count = (i == layerTypes.size() - 1) ? this->numOutputClasses : layerCounts[i];
            if (layerTypes[i] == "Sigmoid") {
                createdLayer = new SigmoidLayer(prevLayerNeurons, count);
            } else {
                throw(layerTypes[i]);
            }

            createdLayer->initializeMatrices(); //initialize the matrices for layer
            layers.push_back(createdLayer);
        }
    }
    catch (int layerTypesSize) {
        std::cout << "Improperly sized initialize layers vectors" << std::endl;
    }
    catch (std::string layerName) {
        std::cout << "Improperly named layer type: " << layerName << std::endl;
    }

}

//single training example, will change input layer activations array to be proper values
void GPUNeuralNetwork::runTrainingExample(std::unique_ptr<std::vector<float> >& exampleInputData, std::unique_ptr<std::vector<float> >& trueLabel, std::vector<Matrix>& gradientCostWeight, std::vector<Matrix>& gradientCostBias) { 
    //set input layer activations
    uint32_t len = this->inputActivations.xDim * this->inputActivations.yDim;
    if (len != (*exampleInputData).size()) {
        std::cout << "error: improperly sized training input" << std::endl;
    }

    for (int i = 0; i < len; i++) {
        this->inputActivations.valuesHost[i] = (*exampleInputData)[i];
    }
    hipMemcpy(this->inputActivations.valuesDevice.get(), this->inputActivations.valuesHost.get(), this->inputActivations.xDim * this->inputActivations.yDim * sizeof(float), hipMemcpyHostToDevice);

    //set true output label vector one hot encoded
    uint32_t len2 = this->trueOutput.xDim * this->trueOutput.yDim;
    if (len2 != (*trueLabel).size()) {
        std::cout << "error: improperly sized true label vector" << std::endl;
    }

    for (int i = 0; i < len2; i++) {
        this->trueOutput.valuesHost[i] = (*trueLabel)[i];
    }
    hipMemcpy(this->trueOutput.valuesDevice.get(), this->trueOutput.valuesHost.get(), this->trueOutput.xDim * this->trueOutput.yDim * sizeof(float), hipMemcpyHostToDevice);

    //forward pass through each layer of network
    for (int i = 0; i < this->layers.size(); i++) {
        if (i == 0) {
            this->layers[i]->forwardPass(this->inputActivations);
        } else {
            this->layers[i]->forwardPass(this->layers[i-1]->outputActivation);
        }
    }

    //obtain cost/loss of current training input and use it to compute error of the final layer

    //backpropagate error through each layer of network to compute input error at each layer, starting with layer L-1
    for (int i = this->layers.size() - 2; i >= 0; i--) {
        this->layers[i]->backprop(this->layers[i+1]->inputError, this->layers[i+1]->weights);
    }

    //Update gradient matrices
}

//Mini batch will call runTrainingExample() on all training inputs in mini batch of size m, use that to perform gradient descent
//inputData is a vector of size m, where each element inputData[m] is a pointer to a vector of one training example's input layer encodings 
void GPUNeuralNetwork::runMiniBatch(std::vector<std::unique_ptr<std::vector<float> > >& inputData, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels, std::vector<Matrix>& gradientCostWeight, std::vector<Matrix>& gradientCostBias) {
    int miniBatchSize = inputData.size();
    if (miniBatchSize == 0) {
        return;
    }

    //reinit running sum of gradients for each layer to 0
    for (int i = 0; i < this->layers.size(); i++) {
        Matrix& w = gradientCostWeight[i];
        Matrix& b = gradientCostBias[i];

        for (int j = 0; j < w.xDim * w.yDim; j++) {
            w.valuesHost[j] = 0.0;
        }

        for (int j = 0; j < b.xDim * b.yDim; j++) {
            b.valuesHost[j] = 0.0;
        }

        hipMemcpy(w.valuesDevice.get(), w.valuesHost.get(), w.xDim * w.yDim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(b.valuesDevice.get(), b.valuesHost.get(), b.xDim * b.yDim * sizeof(float), hipMemcpyHostToDevice);
    }

    
    for (int i = 0; i < miniBatchSize; i++) {
        //std::cout << "[" << (*(inputData[i]))[0] << ", " << (*(inputData[i]))[1] << ", " << (*(inputData[i]))[2] << "]" << std::endl;
        runTrainingExample(inputData[i], trueLabels[i], gradientCostWeight, gradientCostBias); //will update weight and bias gradients
    }

    //obtain average of the gradients after running all training inputs
    //float avgWeightGradient = std::accumulate(gradientCostWeight.begin(), gradientCostWeight.end(), 0.0) / miniBatchSize;
    //float avgBiasGradient = std::accumulate(gradientCostBias.begin(), gradientCostBias.end(), 0.0) / miniBatchSize;

    //Update the weights and biases
}

void GPUNeuralNetwork::randomizeMiniBatches(std::vector<std::unique_ptr<std::vector<float> > >& allTrainingData, std::vector<std::vector<std::unique_ptr<std::vector<float> > > >& miniBatches, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels, std::vector<std::vector<std::unique_ptr<std::vector<float> > > >& trueLabelsBatches, int miniBatchSize, std::default_random_engine& rng) {
    std::vector<int> shuffleIndexes (allTrainingData.size(), 0);
    for (int i = 0; i < shuffleIndexes.size(); i++) {
        shuffleIndexes[i] = i;
    }
    std::shuffle(shuffleIndexes.begin(), shuffleIndexes.end(), rng);
    //std::shuffle(allTrainingData.begin(), allTrainingData.end(), rng);

    for (int i = 0; i < allTrainingData.size(); i++) {
        int currBatchIdx = i / miniBatchSize;
        miniBatches[currBatchIdx].push_back(std::move(allTrainingData[shuffleIndexes[i]]));
        trueLabelsBatches[currBatchIdx].push_back(std::move(trueLabels[shuffleIndexes[i]]));
    }
    //clear all the remaining nullptrs after move
    allTrainingData.clear();
    trueLabels.clear();
}

void GPUNeuralNetwork::trainNetwork(int numEpochs, std::vector<std::unique_ptr<std::vector<float> > >& allTrainingData, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels, int miniBatchSize) {
    int numMiniBatches = (allTrainingData.size() / miniBatchSize) + 1;
    std::vector<std::vector<std::unique_ptr<std::vector<float> > > > miniBatches;
    std::vector<std::vector<std::unique_ptr<std::vector<float> > > > trueLabelsBatches;
    for (int i = 0; i < numMiniBatches; i++) {
        miniBatches.push_back(std::vector<std::unique_ptr<std::vector<float> > >(0));
        trueLabelsBatches.push_back(std::vector<std::unique_ptr<std::vector<float> > >(0));
    }

    auto rng = std::default_random_engine {std::random_device {}()}; //create a reusable instance of default random engine, the () is function call operator overloading after instantiating the random device seed

    //Matrices for each layer of network, keep a running sum of total gradients for each weight and bias in the layer as you go through each training data. Will be averaged after.
    std::vector<Matrix> gradientCostWeight;
    std::vector<Matrix> gradientCostBias;
    for (int i = 0; i < this->layers.size(); i++) {
        gradientCostWeight.emplace_back(Matrix((this->layers[i])->weights.xDim, (this->layers[i])->weights.yDim));
        gradientCostBias.emplace_back(Matrix((this->layers[i])->biases.xDim, (this->layers[i])->biases.yDim));

        Matrix& w = gradientCostWeight[i];
        Matrix& b = gradientCostBias[i];

        w.allocateHostMemory();
        w.allocateCUDAMemory();
        b.allocateHostMemory();
        b.allocateCUDAMemory();
    }
    
    for (int i = 0; i < numEpochs; i++) {
        std::cout << "Beginning Epoch " << i << " of training:" << std::endl;

        //Move everything from mini batch vectors back to allTrainingData vector
        if (i != 0) {
            for (int j = 0; j < numMiniBatches; j++) {
                for (int k = 0; k < miniBatches[j].size(); k++) {
                    allTrainingData.push_back(std::move(miniBatches[j][k]));
                    trueLabels.push_back(std::move(trueLabelsBatches[j][k]));
                }
                //clear all the remaining nullptrs after move
                miniBatches[j].clear();
                trueLabelsBatches[j].clear();
            }

        }

        randomizeMiniBatches(allTrainingData, miniBatches, trueLabels, trueLabelsBatches, miniBatchSize, rng);
        for (int j = 0; j < numMiniBatches; j++) {
            std::cout << "Running Mini Batch " << j << std::endl;
            runMiniBatch(miniBatches[j], trueLabelsBatches[j], gradientCostWeight, gradientCostBias);
        }
        
    }

}