#include "hip/hip_runtime.h"
#include "GPUNeuralNetwork.cuh"
#include "SigmoidLayer.cuh"
#include "Matrix.cuh"
#include <algorithm>
#include <random>
#include <numeric>

GPUNeuralNetwork::GPUNeuralNetwork(std::string costFunc, int inputLayerNeurons, int numOutputClasses, float learningRate) try : costFunction(costFunc), numInputLayerNeurons{inputLayerNeurons}, numOutputClasses{numOutputClasses}, learningRate{learningRate}, inputActivations{1, inputLayerNeurons}, trueOutput{1, numOutputClasses} 
    {
        //allocate host and cuda memory for input layer activations
        inputActivations.allocateHostMemory();
        inputActivations.allocateCUDAMemory();
        //allocate host and cuda memory for true output vector
        trueOutput.allocateHostMemory();
        trueOutput.allocateCUDAMemory();
    }
    catch (std::string type) {
        std::cout << "Invalid cost function: " << type << std::endl;
    }

GPUNeuralNetwork::~GPUNeuralNetwork() { //destructor
    for (int i = 0; i < this->layers.size(); i++) {
        delete this->layers[i];
    }
}

//final layer count will no matter what be same as numOutputClasses, no matter what is given
void GPUNeuralNetwork::initializeLayers(std::vector<std::string>& layerTypes, std::vector<int>& layerCounts) {
    try {
        if (layerTypes.size() != layerCounts.size()) {
            throw(layerTypes.size());
        }

        int prevLayerNeurons = 0;
        for (int i = 0; i < layerTypes.size(); i++) {
            
            if (i == 0) {
                prevLayerNeurons = this->numInputLayerNeurons;
            } else {
                prevLayerNeurons = layerCounts[i-1];
            }

            Layer* createdLayer;
            //final layer count will no matter what be same as numOutputClasses, no matter what is given
            int count = (i == layerTypes.size() - 1) ? this->numOutputClasses : layerCounts[i];
            if (layerTypes[i] == "Sigmoid") {
                createdLayer = new SigmoidLayer(prevLayerNeurons, count);
            } else {
                throw(layerTypes[i]);
            }

            createdLayer->initializeMatrices(); //initialize the matrices for layer
            layers.push_back(createdLayer);
        }
    }
    catch (int layerTypesSize) {
        std::cout << "Improperly sized initialize layers vectors" << std::endl;
    }
    catch (std::string layerName) {
        std::cout << "Improperly named layer type: " << layerName << std::endl;
    }

}

__global__ void costWeightGradientExample(float* error, float* prev_a, float* g_w, int xDim, int yDim) {
    int rowIndex = threadIdx.y + blockDim.y * blockIdx.y;
    int stride_y = blockDim.y * gridDim.y;
    int colIndex = threadIdx.x + blockDim.x + blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;

    for (int i = rowIndex; i < yDim; i = i + stride_y) {
        for (int j = colIndex; j < xDim; j = j + stride_x) {
            g_w[(i * xDim) + j] += error[i] * prev_a[j];
        }
    }


}

__global__ void updateWeights(float* w, float* g_w, int m, float learningRate, int xDim, int yDim) {
    int rowIndex = threadIdx.y + blockDim.y * blockIdx.y;
    int stride_y = blockDim.y * gridDim.y;
    int colIndex = threadIdx.x + blockDim.x + blockIdx.x;
    int stride_x = blockDim.x * gridDim.x;

    for (int i = rowIndex; i < yDim; i = i + stride_y) {
        for (int j = colIndex; j < xDim; j = j + stride_x) {
            w[(i * xDim) + j] = w[(i * xDim) + j] - ((learningRate / (1.0 * m)) * g_w[(i * xDim) + j]);
        }
    }

}

__global__ void updateBiases(float* b, float* g_b, int m, float learningRate, int xDim, int yDim) {
    int rowIndex = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = rowIndex; i < yDim; i = i + stride) {
        b[i] = b[i] - ((learningRate / (1.0 * m)) * g_b[i]);
    }

}

//single training example, will change input layer activations array to be proper values
void GPUNeuralNetwork::runTrainingExample(std::unique_ptr<std::vector<float> >& exampleInputData, std::unique_ptr<std::vector<float> >& trueLabel, std::vector<Matrix>& gradientCostWeight, std::vector<Matrix>& gradientCostBias) { 
    //set input layer activations
    uint32_t len = this->inputActivations.xDim * this->inputActivations.yDim;
    if (len != (*exampleInputData).size()) {
        std::cout << "error: improperly sized training input" << std::endl;
    }

    for (int i = 0; i < len; i++) {
        this->inputActivations.valuesHost[i] = (*exampleInputData)[i];
    }
    hipMemcpy(this->inputActivations.valuesDevice.get(), this->inputActivations.valuesHost.get(), this->inputActivations.xDim * this->inputActivations.yDim * sizeof(float), hipMemcpyHostToDevice);

    //set true output label vector one hot encoded
    uint32_t len2 = this->trueOutput.xDim * this->trueOutput.yDim;
    if (len2 != (*trueLabel).size()) {
        std::cout << "error: improperly sized true label vector" << std::endl;
    }

    for (int i = 0; i < len2; i++) {
        this->trueOutput.valuesHost[i] = (*trueLabel)[i];
    }
    hipMemcpy(this->trueOutput.valuesDevice.get(), this->trueOutput.valuesHost.get(), this->trueOutput.xDim * this->trueOutput.yDim * sizeof(float), hipMemcpyHostToDevice);

    //forward pass through each layer of network
    for (int i = 0; i < this->layers.size(); i++) {
        if (i == 0) {
            this->layers[i]->forwardPass(this->inputActivations);
        } else {
            this->layers[i]->forwardPass(this->layers[i-1]->outputActivation);
        }
    }

    //backpropagate error through each layer of network to compute input error at each layer, update cost w.r.t bias gradient
    float* error;
    float* prev_a;
    float* g_w;

    int num_threadsx = 16; //just set 256 threads per block now; testing to do.
    int num_threadsy = 16;
    int num_blocksx;
    int num_blocksy;
    dim3 blocks;
    dim3 threads = dim3(num_threadsx, num_threadsy); //2d thread dimensions per block

    for (int i = this->layers.size() - 1; i >= 0; i--) {
        if (i == this->layers.size() - 1) { //looking at final layer
            //compute error of the final layer, update cost w.r.t bias gradient
            this->costFunction.getErrorFinalLayer(this->layers[i], this->trueOutput, gradientCostBias[i]);

        } else {
            //backprop error, update cost w.r.t bias gradient
            this->layers[i]->backprop(this->layers[i+1]->inputError, this->layers[i+1]->weights, gradientCostBias[i]);

        }

        //Update cost w.r.t weight gradient matrix
        error = this->layers[i]->inputError.valuesDevice.get();
        prev_a = (i == 0) ? this->inputActivations.valuesDevice.get() : this->layers[i-1]->outputActivation.valuesDevice.get();
        g_w = gradientCostWeight[i].valuesDevice.get();
        //figure out block/grid dimensions:
        num_blocksx = std::ceil((1.0 * gradientCostWeight[i].xDim) / num_threadsx);
        num_blocksy = std::ceil((1.0 * gradientCostWeight[i].yDim) / num_threadsy);
        blocks = dim3(num_blocksx, num_blocksy); //2d block dimensions in grid
        costWeightGradientExample<<<blocks, threads>>>(error, prev_a, g_w, gradientCostWeight[i].xDim, gradientCostWeight[i].yDim); //Update cost w.r.t weight gradient matrix
        hipDeviceSynchronize(); 
        //no need to cudamemcpy to host until later
    }

}

//Mini batch will call runTrainingExample() on all training inputs in mini batch of size m, use that to perform gradient descent
//inputData is a vector of size m, where each element inputData[m] is a pointer to a vector of one training example's input layer encodings 
void GPUNeuralNetwork::runMiniBatch(std::vector<std::unique_ptr<std::vector<float> > >& inputData, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels, std::vector<Matrix>& gradientCostWeight, std::vector<Matrix>& gradientCostBias) {
    int miniBatchSize = inputData.size();
    if (miniBatchSize == 0) {
        return;
    }

    //reinit running sum of gradients for each layer to 0
    for (int i = 0; i < this->layers.size(); i++) {
        Matrix& w = gradientCostWeight[i];
        Matrix& b = gradientCostBias[i];

        for (int j = 0; j < w.xDim * w.yDim; j++) {
            w.valuesHost[j] = 0.0;
        }

        for (int j = 0; j < b.xDim * b.yDim; j++) {
            b.valuesHost[j] = 0.0;
        }

        hipMemcpy(w.valuesDevice.get(), w.valuesHost.get(), w.xDim * w.yDim * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(b.valuesDevice.get(), b.valuesHost.get(), b.xDim * b.yDim * sizeof(float), hipMemcpyHostToDevice);
    }

    
    for (int i = 0; i < miniBatchSize; i++) {
        //std::cout << "[" << (*(inputData[i]))[0] << ", " << (*(inputData[i]))[1] << ", " << (*(inputData[i]))[2] << "]" << std::endl;
        runTrainingExample(inputData[i], trueLabels[i], gradientCostWeight, gradientCostBias); //will update weight and bias gradients
    }

    //obtain average of the gradients after running all training inputs and update all weights and biases
    int num_threadsx = 16; //just set 256 threads per block now; testing to do.
    int num_threadsy = 16;
    dim3 threads; //2d thread dimensions per block
    int num_blocksx;
    int num_blocksy;
    dim3 blocks; //2d block dimensions in grid

    for (int i = 0; i < this->layers.size(); i++) {
        num_threadsx = 16;
        num_blocksx = std::ceil((1.0 * gradientCostWeight[i].xDim) / num_threadsx);
        num_blocksy = std::ceil((1.0 * gradientCostWeight[i].yDim) / num_threadsy);
        threads = dim3(num_threadsx, num_threadsy);
        blocks = dim3(num_blocksx, num_blocksy);

        updateWeights<<<blocks, threads>>>(this->layers[i]->weights.valuesDevice.get(), gradientCostWeight[i].valuesDevice.get(), miniBatchSize, this->learningRate, gradientCostWeight[i].xDim, gradientCostWeight[i].yDim);
        hipDeviceSynchronize(); 
        hipMemcpy(this->layers[i]->weights.valuesHost.get(), this->layers[i]->weights.valuesDevice.get(), this->layers[i]->weights.xDim * this->layers[i]->weights.yDim * sizeof(float), hipMemcpyDeviceToHost);

        num_threadsx = 256;
        num_blocksx = std::ceil((1.0 * gradientCostBias[i].yDim) / num_threadsx);
        threads = dim3(num_threadsx);
        blocks = dim3(num_blocksx);
        updateBiases<<<blocks, threads>>>(this->layers[i]->biases.valuesDevice.get(), gradientCostBias[i].valuesDevice.get(), miniBatchSize, this->learningRate, gradientCostBias[i].xDim, gradientCostBias[i].yDim);
        hipDeviceSynchronize(); 
        hipMemcpy(this->layers[i]->biases.valuesHost.get(), this->layers[i]->biases.valuesDevice.get(), this->layers[i]->biases.xDim * this->layers[i]->biases.yDim * sizeof(float), hipMemcpyDeviceToHost);
    }

}

void GPUNeuralNetwork::randomizeMiniBatches(std::vector<std::unique_ptr<std::vector<float> > >& allTrainingData, std::vector<std::vector<std::unique_ptr<std::vector<float> > > >& miniBatches, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels, std::vector<std::vector<std::unique_ptr<std::vector<float> > > >& trueLabelsBatches, int miniBatchSize, std::default_random_engine& rng) {
    std::vector<int> shuffleIndexes (allTrainingData.size(), 0);
    for (int i = 0; i < shuffleIndexes.size(); i++) {
        shuffleIndexes[i] = i;
    }
    std::shuffle(shuffleIndexes.begin(), shuffleIndexes.end(), rng);
    //std::shuffle(allTrainingData.begin(), allTrainingData.end(), rng);

    for (int i = 0; i < allTrainingData.size(); i++) {
        int currBatchIdx = i / miniBatchSize;
        miniBatches[currBatchIdx].push_back(std::move(allTrainingData[shuffleIndexes[i]]));
        trueLabelsBatches[currBatchIdx].push_back(std::move(trueLabels[shuffleIndexes[i]]));
    }
    //clear all the remaining nullptrs after move
    allTrainingData.clear();
    trueLabels.clear();
}

void GPUNeuralNetwork::trainNetwork(int numEpochs, std::vector<std::unique_ptr<std::vector<float> > >& allTrainingData, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels, std::vector<std::unique_ptr<std::vector<float> > >& allTestingData, std::vector<std::unique_ptr<std::vector<float> > >& testingLabels, int miniBatchSize) {
    int numMiniBatches = (allTrainingData.size() / miniBatchSize) + 1;
    std::vector<std::vector<std::unique_ptr<std::vector<float> > > > miniBatches;
    std::vector<std::vector<std::unique_ptr<std::vector<float> > > > trueLabelsBatches;
    for (int i = 0; i < numMiniBatches; i++) {
        miniBatches.push_back(std::vector<std::unique_ptr<std::vector<float> > >(0));
        trueLabelsBatches.push_back(std::vector<std::unique_ptr<std::vector<float> > >(0));
    }

    auto rng = std::default_random_engine {std::random_device {}()}; //create a reusable instance of default random engine, the () is function call operator overloading after instantiating the random device seed

    //Matrices for each layer of network, keep a running sum of total gradients for each weight and bias in the layer as you go through each training data. Will be averaged after.
    std::vector<Matrix> gradientCostWeight;
    std::vector<Matrix> gradientCostBias;
    for (int i = 0; i < this->layers.size(); i++) {
        gradientCostWeight.emplace_back(Matrix((this->layers[i])->weights.xDim, (this->layers[i])->weights.yDim));
        gradientCostBias.emplace_back(Matrix((this->layers[i])->biases.xDim, (this->layers[i])->biases.yDim));

        Matrix& w = gradientCostWeight[i];
        Matrix& b = gradientCostBias[i];

        w.allocateHostMemory();
        w.allocateCUDAMemory();
        b.allocateHostMemory();
        b.allocateCUDAMemory();
    }
    
    for (int i = 0; i < numEpochs; i++) {
        std::cout << "Beginning Epoch " << i << " of training:" << std::endl;

        //Move everything from mini batch vectors back to allTrainingData vector
        if (i != 0) {
            for (int j = 0; j < numMiniBatches; j++) {
                for (int k = 0; k < miniBatches[j].size(); k++) {
                    allTrainingData.push_back(std::move(miniBatches[j][k]));
                    trueLabels.push_back(std::move(trueLabelsBatches[j][k]));
                }
                //clear all the remaining nullptrs after move
                miniBatches[j].clear();
                trueLabelsBatches[j].clear();
            }

        }

        randomizeMiniBatches(allTrainingData, miniBatches, trueLabels, trueLabelsBatches, miniBatchSize, rng);
        for (int j = 0; j < numMiniBatches; j++) {
            //std::cout << "Running Mini Batch " << j << std::endl;
            runMiniBatch(miniBatches[j], trueLabelsBatches[j], gradientCostWeight, gradientCostBias);
        }

        //At end of epoch, get testing accuracy
        this->testNetwork(allTestingData, testingLabels);
        
    }

}

void GPUNeuralNetwork::testNetwork(std::vector<std::unique_ptr<std::vector<float> > >& testingData, std::vector<std::unique_ptr<std::vector<float> > >& trueLabels) {
    std::cout << "Testing Network..." << std::endl;

    int totalCorrect = 0;
    for (int j = 0; j < testingData.size(); j++) {
        //set input layer activations
        uint32_t len = this->inputActivations.xDim * this->inputActivations.yDim;
        if (len != (*(testingData[j])).size()) {
            std::cout << "error: improperly sized testing input" << std::endl;
        }

        for (int i = 0; i < len; i++) {
            this->inputActivations.valuesHost[i] = (*(testingData[j]))[i];
        }
        hipMemcpy(this->inputActivations.valuesDevice.get(), this->inputActivations.valuesHost.get(), this->inputActivations.xDim * this->inputActivations.yDim * sizeof(float), hipMemcpyHostToDevice);

        //set true output label vector one hot encoded
        uint32_t len2 = this->trueOutput.xDim * this->trueOutput.yDim;
        if (len2 != (*(trueLabels[j])).size()) {
            std::cout << "error: improperly sized true label vector" << std::endl;
        }

        for (int i = 0; i < len2; i++) {
            this->trueOutput.valuesHost[i] = (*(trueLabels[j]))[i];
        }
        hipMemcpy(this->trueOutput.valuesDevice.get(), this->trueOutput.valuesHost.get(), this->trueOutput.xDim * this->trueOutput.yDim * sizeof(float), hipMemcpyHostToDevice);

        //forward pass through each layer of network
        for (int i = 0; i < this->layers.size(); i++) {
            if (i == 0) {
                this->layers[i]->forwardPass(this->inputActivations);
            } else {
                this->layers[i]->forwardPass(this->layers[i-1]->outputActivation);
            }
        }

        //final layer output activations is predicted class
        //std::cout << "TESTING EXAMPLE #" << j << std::endl;
        float* predictedValues = this->layers[this->layers.size() - 1]->outputActivation.valuesHost.get();
        std::vector<float>& actualValues = *(trueLabels[j]);

        int predictedLabel = 0;
        float predictedLabelValue = -1.0;
        for (int i = 0; i < actualValues.size(); i++) {
            if (predictedValues[i] > predictedLabelValue) {
                predictedLabelValue = predictedValues[i];
                predictedLabel = i;
            }
        }

        int actualLabel = 0;
        for (int i = 0; i < actualValues.size(); i++) {
            if (actualValues[i] == 1.0) {
                actualLabel = i;
                break;
            }

        }

        if (predictedLabel == actualLabel) {
            totalCorrect++;
        }

        // std::cout << "predicted: [";
        // for (int i = 0; i < actualValues.size(); i++) {
        //     if (i == actualValues.size() - 1) {
        //         std::cout << predictedValues[i] << "]" << std::endl;
        //     } else {
        //         std::cout << predictedValues[i] << ", ";
        //     }
        // }

        // std::cout << "actual: [";
        // for (int i = 0; i < actualValues.size(); i++) {
        //     if (i == actualValues.size() - 1) {
        //         std::cout << actualValues[i] << "]" << std::endl;
        //     } else {
        //         std::cout << actualValues[i] << ", ";
        //     }
        // }

    }

    float accuracy = (1.0 * totalCorrect) / testingData.size();
    std::cout << "Total testing accuracy:" << std::endl;
    std::cout << accuracy << std::endl;

}